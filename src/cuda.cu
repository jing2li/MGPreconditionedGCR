#include "hip/hip_runtime.h"
//
// Created by jingjingli on 07/05/24.
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <utility>
#include "Operator.h"

template <typename num_type, typename coarse_num_type>
__global__ void fill_triplet(std::pair<Operator<coarse_num_type>*, std::pair<coarse_num_type, coarse_num_type>>* triplets, Mesh<num_type> mesh)
{

}